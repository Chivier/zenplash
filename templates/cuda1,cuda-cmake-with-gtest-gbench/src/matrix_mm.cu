#include <hipblas.h>
#include "deepgreen/matrix.hh"
#include "deepgreen/matrix_mm.cuh"

namespace deepgreen {

void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
  int lda = m,ldb = k,ldc = m;
  const float alf = 1;
  const float bet = 0;
  const float *alpha = &alf;
  const float *beta = &bet;

  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Do the actual multiplication
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

  // Destroy the handle
  hipblasDestroy(handle);
}

auto cuda_mm(matrix<float> const& a, matrix<float> const& b) -> matrix<float> {
  size_t const m = a.rows(), n = b.cols(), k = a.cols();
  size_t const c_bytes = m * n * sizeof(float);
  float* elems = (float*)std::malloc(c_bytes);

  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, a.bytes());
  hipMalloc(&d_B, b.bytes());
  hipMalloc(&d_C, c_bytes);

  hipMemcpy(d_A, a.data(), a.bytes(), hipMemcpyHostToDevice);
  hipMemcpy(d_B, b.data(), b.bytes(), hipMemcpyHostToDevice);

  gpu_blas_mmul(d_A, d_B, d_C, m, k, n);

  hipMemcpy(elems, d_C, c_bytes, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return matrix<float>(m, n, elems);
}

} /* end namespace deepgreen */
